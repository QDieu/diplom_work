#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <cstdio>
#include <fstream>
#include <vector>
#include <chrono>
#include <ctime>

#define GravConst 6.674e-11
#define THREAD_COUNT 128
#define EPS 1e-6

__device__ constexpr float sqr(float x) { return x * x; }

__host__ void cuAssert(hipError_t err, std::string msg) {
    if (err != hipSuccess) {
        std::cerr << msg << " " << err << std::endl;
        exit(1);
    }
}

__device__ void calcForces(float4 bi, float4 bj, float3& fi) {
    float3 r;
    r.x = bj.x - bi.x;
    r.y = bj.y - bi.y;
    r.z = bj.z - bi.z;
    float dist = sqrt(r.x * r.x + r.y * r.y + r.z * r.z);

    if (dist < 1e-6) dist = 1e-6;

    float F = (GravConst * bi.w * bj.w) / sqr(dist);
    fi.x += F * r.x / dist;
    fi.y += F * r.y / dist;
    fi.z += F * r.z / dist;
}

__global__ void nextStep(float4* pos, float3* v, float3* a, float4* posNew, float3* vNew, float3* aNew, int size, int dt) {
    __shared__ float4 sh_pos[THREAD_COUNT];
    float4 myPosition;
    float3 res = { 0.0f, 0.0f, 0.0f };
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    myPosition = pos[idx];

    //���������� ����� � ������ ������
    if (idx < size)
    {
        for (int i = 0; i < gridDim.x; i++) {
            if (i * blockDim.x + threadIdx.x < size) {
                sh_pos[threadIdx.x] = pos[i * blockDim.x + threadIdx.x];
            }
            __syncthreads();
            for (int k = 0; k < blockDim.x; k++) {
                if (idx != (i * blockDim.x + k) && i*blockDim.x + k < size) calcForces(myPosition, sh_pos[k], res);
            }
            __syncthreads();
        }

        posNew[idx].x = myPosition.x + v[idx].x * dt + (a[idx].x * sqr(dt)) / 2;
        posNew[idx].y = myPosition.y + v[idx].y * dt + (a[idx].y * sqr(dt)) / 2;
        posNew[idx].z = myPosition.z + v[idx].z * dt + (a[idx].z * sqr(dt)) / 2;

        posNew[idx].w = pos[idx].w;

        vNew[idx].x = v[idx].x + a[idx].x * dt;
        vNew[idx].y = v[idx].y + a[idx].y * dt;
        vNew[idx].z = v[idx].z + a[idx].z * dt;

        aNew[idx].x = res.x / pos[idx].w;
        aNew[idx].y = res.y / pos[idx].w;
        aNew[idx].z = res.z / pos[idx].w;
    }
}

void readPointsData(const std::string& name, float4* point, int& size) {
    std::ifstream infile(name);

    for (int i = 0; i < size; i++) {
        float x, y, z, m;
        infile >> x >> y >> z >> m;
        point[i].x = x;
        point[i].y = y;
        point[i].z = z;
        point[i].w = m;
    }
}


void readData(const std::string& name, int& size, int& iterations, int& dt) {

    std::ifstream in(name);
    in >> size >> iterations >> dt;
}

void writeFile(std::ofstream& outfile, float4* point, int& size) {

    for (int i = 0; i < size; i++) {
        outfile << point[i].x << ' ' << point[i].y << ' ' << point[i].z << ' ' << point[i].w<< "\t\t";
    }
    outfile << "\n";

}

//Генератор случайных точек
    //void genPoints() {
    //    srand(time(NULL));
    //    std::ofstream out("inputDataPoint.txt", std::ios::trunc);
    //    for (int i = 0; i < 2000; i++) {
    //        out << 0.001 * (rand() % 2001 - 1000) << " " <<  0.001 * (rand() % 2001 - 1000) << " " << 0.001 * (rand() % 2001 - 1000) << " " << 0.01 * (rand() % 101) + 0.01 << std::endl;
    //    }
    //    out.close();
    //}

int main()
{
    std::ofstream outfile("output.txt", std::ios::trunc);
    int size, iterations, dt;

    readData("Data.txt", size, iterations, dt);
    
    //������� ��� ����� �� �����
    float4* host_pos = (float4*)malloc(sizeof(float4) * size);
    float3* host_v = (float3*)malloc(sizeof(float3) * size);
    float3* host_a = (float3*)malloc(sizeof(float3) * size);
    memset(host_v, 0, size * sizeof(float3));
    memset(host_a, 0, size * sizeof(float3));

    readPointsData("inputDataPoint.txt", host_pos, size);

    //host_v[1].y = -0.0002;
    
    //������ � ������� �� ��������
    float4* cuArr_pos = NULL;
    float3* cuArr_v = NULL;
    float3* cuArr_a = NULL;

    float4* cuResArr_pos = NULL;
    float3* cuResArr_v = NULL;
    float3* cuResArr_a = NULL;

    cuAssert(hipMalloc((void**)&cuArr_pos, size * sizeof(float4)), "CudaMalloc cuArr_pos");
    cuAssert(hipMalloc((void**)&cuArr_v, size * sizeof(float3)), "CudaMalloc cuArr_v");
    cuAssert(hipMalloc((void**)&cuArr_a, size * sizeof(float3)), "CudaMalloc cuArr_a");

    cuAssert(hipMalloc((void**)&cuResArr_pos, size * sizeof(float4)), "CudaMalloc cuResArr_pos");
    cuAssert(hipMalloc((void**)&cuResArr_v, size * sizeof(float3)), "CudaMalloc cuResArr_v");
    cuAssert(hipMalloc((void**)&cuResArr_a, size * sizeof(float3)), "CudaMalloc cuResArr_a");

    cuAssert(hipMemcpy(cuArr_pos, host_pos, size * sizeof(float4), hipMemcpyHostToDevice), "cudaMemcpy_pos");
    cuAssert(hipMemcpy(cuArr_v, host_v, size * sizeof(float3), hipMemcpyHostToDevice), "cudaMemcpy_v");
    cuAssert(hipMemcpy(cuArr_a, host_a, size * sizeof(float3), hipMemcpyHostToDevice), "cudaMemcpy_a");

    dim3 gridDim, blockDim;
    blockDim.x = THREAD_COUNT;
    gridDim.x = (size + blockDim.x - 1) / blockDim.x;

    auto now = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < iterations; i++) {
        //std::cerr << "Number iterations : " << i << std::endl;
        if (i % 2 == 0) {
            nextStep << < gridDim, blockDim >> > (cuArr_pos, cuArr_v, cuArr_a, cuResArr_pos, cuResArr_v, cuResArr_a, size, dt);
            cuAssert(hipDeviceSynchronize(), "CudaSyncronize");
            cuAssert(hipMemcpy(host_pos, cuResArr_pos, size * sizeof(float4), hipMemcpyDeviceToHost), "hipMemcpy DTH_pos");
            cuAssert(hipMemcpy(host_v, cuResArr_v, size * sizeof(float3), hipMemcpyDeviceToHost), "hipMemcpy DTH_v");
            cuAssert(hipMemcpy(host_a, cuResArr_a, size * sizeof(float3), hipMemcpyDeviceToHost), "hipMemcpy DTH_a");
        }
        else {
            nextStep << < gridDim, blockDim >> > (cuResArr_pos, cuResArr_v, cuResArr_a, cuArr_pos, cuArr_v, cuArr_a, size, dt);
            cuAssert(hipDeviceSynchronize(), "CudaSyncronize");
            cuAssert(hipMemcpy(host_pos, cuArr_pos, size * sizeof(float4), hipMemcpyDeviceToHost), "hipMemcpy DTH_pos");
            cuAssert(hipMemcpy(host_v, cuArr_v, size * sizeof(float3), hipMemcpyDeviceToHost), "hipMemcpy DTH_v");
            cuAssert(hipMemcpy(host_a, cuArr_a, size * sizeof(float3), hipMemcpyDeviceToHost), "hipMemcpy DTH_a");
        }
        

        writeFile(outfile, host_pos ,size);

        /*if (i != 0) {
            float4* tmpArr_pos = cuArr_pos;
            cuArr_pos = cuResArr_pos;
            cuResArr_pos = tmpArr_pos;
            float3* tmpArr_a= cuArr_a;
            cuArr_a = cuResArr_a;
            cuResArr_a = tmpArr_a;
            float3* tmpArr_v = cuArr_v;
            cuArr_v = cuResArr_v;
            cuResArr_v = tmpArr_v;
        }*/
    }

    auto elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - now);
    std::cerr << "Time : " << elapsed.count() << "ms.\n";

    hipFree(cuResArr_a);
    hipFree(cuResArr_v);
    hipFree(cuResArr_pos);

    hipFree(cuArr_a);
    hipFree(cuArr_v);
    hipFree(cuArr_pos);

    free(host_a);
    free(host_v);
    free(host_pos);

    outfile.close();
    /*genPoints();*/

    return 0;
}